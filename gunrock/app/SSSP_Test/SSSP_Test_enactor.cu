#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * Template_enactor.cuh
 *
 * @brief Template Problem Enactor
 */

#pragma once

#include <gunrock/app/enactor_base.cuh>
#include <gunrock/app/enactor_iteration.cuh>
#include <gunrock/app/enactor_loop.cuh>
#include <gunrock/app/SSSP_Test/SSSP_Test_problem.cuh>
#include <gunrock/oprtr/oprtr.cuh>

namespace gunrock {
namespace app {
namespace SSSP_Test {

/**
 * @brief Speciflying parameters for SSSP Enactor
 * @param parameters The util::Parameter<...> structure holding all parameter info
 * \return hipError_t error message(s), if any
 */
hipError_t UseParameters_enactor(util::Parameters &parameters)
{
    hipError_t retval = hipSuccess;
    GUARD_CU(app::UseParameters_enactor(parameters));

    return retval;
}

/**
 * @brief defination of SSSP iteration loop
 * @tparam EnactorT Type of enactor
 */
template <typename EnactorT>
struct TemplateIterationLoop : public IterationLoopBase
    <EnactorT, Use_FullQ | Push
    | (((EnactorT::Problem::FLAG & Mark_Predecessors) != 0) ?
    Update_Predecessors : 0x0)
    >
{
    typedef typename EnactorT::VertexT VertexT;
    typedef typename EnactorT::SizeT   SizeT;
    typedef typename EnactorT::ValueT  ValueT;
    typedef typename EnactorT::Problem::GraphT::CsrT CsrT;
    typedef typename EnactorT::Problem::GraphT::GpT  GpT;
    typedef IterationLoopBase
        <EnactorT, Use_FullQ | Push
        | (((EnactorT::Problem::FLAG & Mark_Predecessors) != 0) ?
        Update_Predecessors : 0x0)
        > BaseIterationLoop;

    TemplateIterationLoop() : BaseIterationLoop() {}

    /**
     * @brief Core computation of sssp, one iteration
     * @param[in] peer_ Which GPU peers to work on, 0 means local
     * \return hipError_t error message(s), if any
     */
    hipError_t Core(int peer_ = 0)
    {
        // Data alias the enactor works on
        auto         &data_slice         =   this -> enactor ->
            problem -> data_slices[this -> gpu_num][0];
        auto         &enactor_slice      =   this -> enactor ->
            enactor_slices[this -> gpu_num * this -> enactor -> num_gpus + peer_];
        auto         &enactor_stats      =   enactor_slice.enactor_stats;
        auto         &graph              =   data_slice.sub_graph[0];
        auto         &frontier           =   enactor_slice.frontier;
        auto         &oprtr_parameters   =   enactor_slice.oprtr_parameters;
        auto         &retval             =   enactor_stats.retval;
        auto         &iteration          =   enactor_stats.iteration;
        auto         &distances          =   data_slice.distances;
        auto         &weights            =   graph.CsrT::edge_values;
        auto         &original_vertices  =   graph.GpT::original_vertices;
        auto         &preds              =   data_slice.preds;
        auto         &labels             =   data_slice.labels;

        // The advance operation
        auto advance_op = [
          distances, weights, original_vertices, preds
        ] __host__ __device__ (
            const VertexT &src, VertexT &dest, const SizeT &edge_id,
            const VertexT &input_item, const SizeT &input_pos,
            SizeT &output_pos) -> bool
        {
            ValueT src_distance = distances[src];
            ValueT edge_weight  = weights[edge_id];
            ValueT new_distance = src_distance + edge_weight;

            // Check if the destination node has been claimed as someone's child
            ValueT old_distance = atomicMin(distances + dest, new_distance);

            if (new_distance < old_distance)
            { // keep dest in the output frontier if distance has been updated
                if (!preds.IsEmpty()) preds[dest] = src;
                return true;
            }
            return false;
        };

        // The filter operation
        auto filter_op = [
         labels, iteration
        ] __host__ __device__ (
            const VertexT &src, VertexT &dest, const SizeT &edge_id,
            const VertexT &input_item, const SizeT &input_pos,
            SizeT &output_pos) -> bool
        {
            if (!util::isValid(dest)) return false;
            if (labels[dest] == iteration) return false;
            labels[dest] = iteration;
            return true;
        };

        oprtr_parameters.label = iteration + 1;
        GUARD_CU(oprtr::Advance<oprtr::OprtrType_V2V>(
            graph.csr(), frontier.V_Q(), frontier.Next_V_Q(),
            oprtr_parameters, advance_op, filter_op));

        if (oprtr_parameters.advance_mode != "LB_CULL" &&
            oprtr_parameters.advance_mode != "LB_LIGHT_CULL")
        {
            frontier.queue_reset = false;
            // Call the filter operator, using the filter operation
            GUARD_CU(oprtr::Filter<oprtr::OprtrType_V2V>(
                graph.csr(), frontier.V_Q(), frontier.Next_V_Q(),
                oprtr_parameters, filter_op));
        }

        // Get back the resulted frontier length
        GUARD_CU(frontier.work_progress.GetQueueLength(
            frontier.queue_index, frontier.queue_length,
            false, oprtr_parameters.stream, true));

        return retval;
    }

    /**
     * @brief Routine to combine received data and local data
     * @tparam NUM_VERTEX_ASSOCIATES Number of data associated with each transmition item, typed VertexT
     * @tparam NUM_VALUE__ASSOCIATES Number of data associated with each transmition item, typed ValueT
     * @param  received_length The numver of transmition items received
     * @param[in] peer_ which peer GPU the data came from
     * \return hipError_t error message(s), if any
     */
    template <
        int NUM_VERTEX_ASSOCIATES,
        int NUM_VALUE__ASSOCIATES>
    hipError_t ExpandIncoming(SizeT &received_length, int peer_)
    {
        auto         &data_slice         =   this -> enactor ->
            problem -> data_slices[this -> gpu_num][0];
        auto         &enactor_slice      =   this -> enactor ->
            enactor_slices[this -> gpu_num * this -> enactor -> num_gpus + peer_];
        //auto iteration = enactor_slice.enactor_stats.iteration;
        auto         &distances          =   data_slice.distances;

        auto expand_op = [
         distances
        ] __host__ __device__(
            VertexT &key, const SizeT &in_pos,
            VertexT *vertex_associate_ins,
            ValueT  *value__associate_ins) -> bool
        {
            ValueT in_val  = value__associate_ins[in_pos];
            ValueT old_val = atomicMin(distances + key, in_val);
            if (old_val <= in_val)
                return false;
            return true;
        };

        hipError_t retval = BaseIterationLoop:: template ExpandIncomingBase
            <NUM_VERTEX_ASSOCIATES, NUM_VALUE__ASSOCIATES>
            (received_length, peer_, expand_op);
        return retval;
    }
}; // end of SSSPIteration

/**
 * @brief Template enactor class.
 * @tparam _Problem Problem type we process on
 * @tparam ARRAY_FLAG Flags for util::Array1D used in the enactor
 * @tparam cudaHostRegisterFlag Flags for util::Array1D used in the enactor
 */
template <
    typename _Problem,
    util::ArrayFlag ARRAY_FLAG = util::ARRAY_NONE,
    unsigned int cudaHostRegisterFlag = hipHostRegisterDefault>
class Enactor :
    public EnactorBase<
        typename _Problem::GraphT,
        typename _Problem::GraphT::VertexT,
        typename _Problem::GraphT::ValueT,
        ARRAY_FLAG, cudaHostRegisterFlag>
{
public:
    typedef _Problem                   Problem ;
    typedef typename Problem::SizeT    SizeT   ;
    typedef typename Problem::VertexT  VertexT ;
    typedef typename Problem::GraphT   GraphT  ;
    typedef typename GraphT::VertexT   LabelT  ;
    typedef typename GraphT::ValueT    ValueT  ;
    typedef EnactorBase<GraphT , LabelT, ValueT, ARRAY_FLAG, cudaHostRegisterFlag>
        BaseEnactor;
    typedef Enactor<Problem, ARRAY_FLAG, cudaHostRegisterFlag>
        EnactorT;
    typedef TemplateIterationLoop<EnactorT> IterationT;

    Problem     *problem   ;
    IterationT  *iterations;

    /**
     * @brief SSSPEnactor constructor
     */
    Enactor() :
        BaseEnactor("Template"),
        problem    (NULL  )
    {
        this -> max_num_vertex_associates = 0;
        this -> max_num_value__associates = 1;
    }

    /**
     * @brief SSSPEnactor destructor
     */
    virtual ~Enactor()
    {
        //Release();
    }

    /*
     * @brief Releasing allocated memory space
     * @param target The location to release memory from
     * \return hipError_t error message(s), if any
     */
    hipError_t Release(util::Location target = util::LOCATION_ALL)
    {
        hipError_t retval = hipSuccess;
        GUARD_CU(BaseEnactor::Release(target));
        delete []iterations; iterations = NULL;
        problem = NULL;
        return retval;
    }

    /**
     * \addtogroup PublicInterface
     * @{
     */

    /**
     * @brief Initialize the problem.
     * @param[in] parameters Running parameters.
     * @param[in] problem The problem object.
     * @param[in] target Target location of data
     * \return hipError_t error message(s), if any
     */
    hipError_t Init(
        //util::Parameters &parameters,
        Problem          &problem,
        util::Location    target = util::DEVICE)
    {
        hipError_t retval = hipSuccess;
        this->problem = &problem;

        // Lazy initialization
        GUARD_CU(BaseEnactor::Init(
            problem, Enactor_None,
            2, NULL,
            target, false));
        for (int gpu = 0; gpu < this -> num_gpus; gpu ++)
        {
            GUARD_CU(util::SetDevice(this -> gpu_idx[gpu]));
            auto &enactor_slice
                = this -> enactor_slices[gpu * this -> num_gpus + 0];
            auto &graph = problem.sub_graphs[gpu];
            GUARD_CU(enactor_slice.frontier.Allocate(
                graph.nodes, graph.edges, this -> queue_factors));
        }

        iterations = new IterationT[this -> num_gpus];
        for (int gpu = 0; gpu < this -> num_gpus; gpu ++)
        {
            GUARD_CU(iterations[gpu].Init(this, gpu));
        }

        GUARD_CU(this -> Init_Threads(this,
            (CUT_THREADROUTINE)&(GunrockThread<EnactorT>)));
        return retval;
    }

    /**
      * @brief one run of sssp, to be called within GunrockThread
      * @param thread_data Data for the CPU thread
      * \return hipError_t error message(s), if any
      */
    hipError_t Run(ThreadSlice &thread_data)
    {
        gunrock::app::Iteration_Loop<
            0, 1,
            IterationT>(
            thread_data, iterations[thread_data.thread_num]);
        return hipSuccess;
    }

    /**
     * @brief Reset enactor
     * @param[in] src Source node to start primitive.
     * @param[in] target Target location of data
     * \return hipError_t error message(s), if any
     */
    hipError_t Reset(
        VertexT src,
        util::Location target = util::DEVICE)
    {
        typedef typename GraphT::GpT GpT;
        hipError_t retval = hipSuccess;
        GUARD_CU(BaseEnactor::Reset(target));

        for (int gpu = 0; gpu < this->num_gpus; gpu++)
        {
           if ((this->num_gpus == 1) ||
                (gpu == this->problem->org_graph->GpT::partition_table[src]))
           {
               this -> thread_slices[gpu].init_size = 1;
               for (int peer_ = 0; peer_ < this -> num_gpus; peer_++)
               {
                   auto &frontier = this ->
                       enactor_slices[gpu * this -> num_gpus + peer_].frontier;
                   frontier.queue_length = (peer_ == 0) ? 1 : 0;
                   if (peer_ == 0)
                   {
                       GUARD_CU(frontier.V_Q() -> ForEach(
                           [src]__host__ __device__ (VertexT &v)
                       {
                           v = src;
                       }, 1, target, 0));
                   }
               }
           }

           else {
                this -> thread_slices[gpu].init_size = 0;
                for (int peer_ = 0; peer_ < this -> num_gpus; peer_++)
                {
                    this -> enactor_slices[gpu * this -> num_gpus + peer_]
                        .frontier.queue_length = 0;
                }
           }
        }
        GUARD_CU(BaseEnactor::Sync());
        return retval;
    }

    /**
     * @brief Enacts a SSSP computing on the specified graph.
     * @param[in] src Source node to start primitive.
     * \return hipError_t error message(s), if any
     */
    hipError_t Enact(
        VertexT src
        )
    {
        hipError_t  retval     = hipSuccess;
        GUARD_CU(this -> Run_Threads(this));
        util::PrintMsg("GPU Template Done.", this -> flag & Debug);
        return retval;
    }

    /** @} */
};

} // namespace Template
} // namespace app
} // namespace gunrock

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
